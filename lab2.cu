
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>


#define N 96 // Size of the arrays


__global__ void GPUatomicmax(int *input, int *max2, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // totale index = nummer van blok waar we nu inzitten x aantal blokken + nummer van de thread
    int stride = blockDim.x*gridDim.x;

    __shared__ int max;
    max = 0;

    for(int i = tid; i<size; i+=stride){
        atomicMax(&max,  input[i]);
    }

    atomicMax(max2, max);
}

__global__ void GPUreductionmax(int *input, int *max3, int size){
    //int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //int stride = blockDim.x*gridDim.x;

    //int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    //sdata[tid] = (i < size) ? input[i] : INT_MIN;
    __syncthreads();

    //for (unsigned int s = blockDim.x; s > 0; s >>= 1) {
    //    if (tid < s) {
    //        input[tid] = max(input[tid], input[tid + s]);
    //    }
    //    __syncthreads();
    //}
    for (int s = 1; s < size/2 ; s++) {
        if (tid < s) {
            input[s*tid] = max(input[s*tid], input[s*(tid + 1)]);
        }
        __syncthreads();
    }
     *max3 = input[0];
    //*max3 = 5;
     //relatie tussen de verschillende -> bij de eerste is *2 en +1, volgende is *4 en +2 , volgende ...
     
}





// CPU code
int main() {
    int inputArray[N];
    for (int i=0; i<N; i++){
        inputArray[i] = rand();
    }
    int *d_inputArray;
    hipMalloc(&d_inputArray, N * sizeof(int)); 
    hipMemcpy(d_inputArray, inputArray, N * sizeof(int), hipMemcpyHostToDevice); // cpu memory to GPU memory

    // Define grid and block size
    int blockSize = 256;  // aantal threads in block
    int gridSize = (N + blockSize - 1) / blockSize;


    //CPU
    int maxcpu = 0;
    const auto start = std::chrono::steady_clock::now();

    for ( int i = 0; i<N; i++) {
        for(int j=i+1; j<N; j++) {
            if(inputArray[i]>inputArray[j] & inputArray[i]>maxcpu){
                maxcpu = inputArray[i];
            }
        }
    }

    const auto end = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed_seconds_cpu{end - start};

    //GPU atomic
    int maxgpu1 = 0;
    int *d_maxgpu1;
    hipMalloc(&d_maxgpu1, sizeof(int)); 
    const auto startgpu1 = std::chrono::high_resolution_clock::now();

    GPUatomicmax<<<gridSize, blockSize>>>(d_inputArray, d_maxgpu1, N);  // geef pointers naar arrays mee
    hipMemcpy(&maxgpu1, d_maxgpu1, sizeof(int), hipMemcpyDeviceToHost); // cpu memory to GPU memory

    //cudaFree(d_inputArray);
    hipFree(d_maxgpu1);
    
    const auto endgpu1 = std::chrono::high_resolution_clock::now();
    const std::chrono::duration<double> elapsed_seconds_gpu{endgpu1 - startgpu1};


    //GPU reduction
    int maxgpu2 = 0;
    int *d_maxgpu2;
    hipMalloc(&d_maxgpu2, sizeof(int)); 
    const auto startgpu2 = std::chrono::high_resolution_clock::now();

    GPUreductionmax<<<gridSize, blockSize>>>(d_inputArray, d_maxgpu2, N);

    hipDeviceSynchronize();

    hipMemcpy(&maxgpu2, d_maxgpu2, sizeof(int), hipMemcpyDeviceToHost);
    
    hipFree(d_inputArray);
    hipFree(d_maxgpu2);

    const auto endgpu2 = std::chrono::high_resolution_clock::now();
    const std::chrono::duration<double> elapsed_seconds_gpu2{endgpu2 - startgpu2};

    // Print the result (standaard cpp stuff)
    std::cout << "Input Array: ";
    for (int i = 0; i < N; i++) {
        std::cout << inputArray[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "CPU" << std::endl;
    std::cout << "Max: " << maxcpu << std::endl;
    std::cout << elapsed_seconds_cpu.count() << "\n";

    std::cout << "GPU atomic" << std::endl;
    std::cout << "Max: " << maxgpu1 << std::endl;
    std::cout << elapsed_seconds_gpu.count() << "\n";

    std::cout << "GPU reduction" << std::endl;
    std::cout << "Max: " << maxgpu2 << std::endl;

    std::cout << elapsed_seconds_gpu2.count() << "\n";

    return 0;
}