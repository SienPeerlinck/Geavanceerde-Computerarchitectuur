
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <bitset>

using namespace std;

#define width 384
#define height 288
#define gridSize 1
#define blockSize 96

uint8_t* get_image_array() {
    FILE* imageFile = fopen("./1x2a-night-000.png", "rb");
    if (imageFile == NULL) {
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }
    uint8_t* image_array = (uint8_t*)malloc(width * height * sizeof(uint8_t));
    fread(image_array, sizeof(uint8_t), width * height, imageFile);
    fclose(imageFile);
    return image_array;
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void testKernel(uint8_t* image_array, int* output_array) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < width * height; i += stride) {
        output_array[i * 35] = 5; // Set a known value
    }
}

int main(void) {
    uint8_t* image_array = get_image_array();
    int* output_array = (int*)malloc(width * height * 35 * sizeof(int));

    uint8_t* d_image_array;
    hipError_t err;
    err = hipMalloc(&d_image_array, width * height * sizeof(uint8_t));
    checkCudaError(err, "Failed to allocate device memory for image array");

    err = hipMemcpy(d_image_array, image_array, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
    checkCudaError(err, "Failed to copy image array to device");

    int* d_output_array;
    err = hipMalloc(&d_output_array, width * height * 35 * sizeof(int));
    checkCudaError(err, "Failed to allocate device memory for output array");

    const auto start = std::chrono::high_resolution_clock::now();
    testKernel<<<gridSize, blockSize>>>(d_image_array, d_output_array);
    err = hipGetLastError();
    checkCudaError(err, "Kernel launch failed");

    err = hipMemcpy(output_array, d_output_array, width * height * 35 * sizeof(int), hipMemcpyDeviceToHost);
    checkCudaError(err, "Failed to copy output array from device");

    hipDeviceSynchronize();
    const auto end = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < width * height; i++) {
        std::cout << output_array[i * 35] << " " << i << std::endl;
    }

    hipFree(d_image_array);
    hipFree(d_output_array);
    free(image_array);
    free(output_array);

    return 0;
}
